#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define HANDLE_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

std::string load_ptx(const std::string& filename) {
    std::ifstream file(filename);
    if (!file) {
        std::cerr << "Cannot open PTX file: " << filename << std::endl;
        exit(1);
    }
    return std::string(std::istreambuf_iterator<char>(file), std::istreambuf_iterator<char>());
}

int main() {
  int N = 1024;
  float *a_dev, *b_dev, *c_dev;

  HANDLE_ERROR(hipMalloc(&a_dev, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc(&b_dev, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc(&c_dev, N * sizeof(float)));

  std::vector<float> a(N, 1.0f), b(N, 2.0f), c(N, 0.0f);
  HANDLE_ERROR(
      hipMemcpy(a_dev, a.data(), N * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(b_dev, b.data(), N * sizeof(float), hipMemcpyHostToDevice));

  std::string ptx = load_ptx("./vadd.ptx");
  std::cout << ptx << std::endl;
  hipModule_t cuModule;
  hipFunction_t cuFunction;
  hipInit(0);
  hipDevice_t cuDevice;
  hipDeviceGet(&cuDevice, 0);
  hipCtx_t cuContext;
  hipCtxCreate(&cuContext, 0, cuDevice);
  hipError_t res = hipModuleLoadDataEx(&cuModule, ptx.c_str(), 0, 0, 0);
  if (res != hipSuccess) {
    std::cerr << "Failed to load module: " << res << std::endl;
    exit(1);
  }
  hipModuleGetFunction(&cuFunction, cuModule, "add_kernel");

  void *args[] = {&a_dev, &b_dev, &c_dev, reinterpret_cast<void *>(&N)};
  int threadsPerBlock = 128;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  hipModuleLaunchKernel(cuFunction, blocksPerGrid, 1, 1, threadsPerBlock, 1, 1, 0,
                 NULL, args, NULL);
  hipCtxSynchronize();

  HANDLE_ERROR(
      hipMemcpy(c.data(), c_dev, N * sizeof(float), hipMemcpyDeviceToHost));

  for (int i = 0; i < N; ++i) {
    std::cout << "c[" << i << "] = " << c[i] << std::endl;
  }

  hipFree(a_dev);
  hipFree(b_dev);
  hipFree(c_dev);
  hipModuleUnload(cuModule);
  hipCtxDestroy(cuContext);

  return 0;
}
